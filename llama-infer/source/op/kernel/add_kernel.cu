#include "hip/hip_runtime.h"

#include "kernel/add.h"

__global__ void add_kernel(
    const float* input1,
    const float* input2,
    const int size,
    float* output
){
    int idx = threadIdx.x + blockIdx.x * gridDim.x;
    if(idx<size){
        output[idx] = input1[idx] + input2[idx];
    }
}


void add_gpu(
    const float* input1,
    const float* input2,
    const int size,
    float* output
){
    dim3 blockDim(32*32);
    dim3 gridDim((size-blockDim.x+1)/(blockDim.x));
    add_kernel<<<gridDim, blockDim>>>(input1, input2, size, output);
}